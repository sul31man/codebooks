#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void basic_kernel(int* A, int n ){


    int idx = blockDim * blockIdx + threadIdx;

    if (idx < n){

        A[idx] = A[idx] + 1;

    }

    //super basic kernel that we will use to set an example of a cuda graph
}

void checkCudaError(hipError_t error){

    if (error != hipSuccess){

        printf("brav");
    }
}

int main(){

    int* A; //intialise the pointe to our array
    int* A_d; 
    int n = 1024; 

    hipGraph_t graph ;
    hipGraphExec_t graph_exec; 

    hipStream_t stream; 
    hipStreamCreate(&stream);

    cudaHostAllocate(&A, n*sizeof(int) , hipHostMallocDefault);//this frees up the memory efficiently for our array and assigns it to A
    hipMalloc(&A_d, n*sizeof(int));
    //now lets assign some random numbers to the array;

    for(int i=0; i < n; i++){

        A[i] = i;
    }

    
    
    int threadPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1)/(threadsPerBlock) ; 

    checkCudaError(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    hipMemcpyAsync(A_d, A, n*sizeof(int), cudaHostToDevice, stream);

    basic_kernel<<<blocks, threadsPerBlock, 0, stream>>>(A_d, n);

    hipMemcpyAsync(A, A_d, n*sizeof(int), cudaDeviceToHost, stream);

    checkCudaError(hipStreamEndCapture(stream, &graph));

    checkCudaError(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));

    hipGraphLaunch(graph_exec, stream);


    hipStreamBeginCapture(stream, cudaStreamCaptureModeGLobal);
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0);
    hipGraphLaunch(graph_exec, stream);


    

    



}