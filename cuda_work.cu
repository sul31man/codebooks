#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel function for vector addition
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 1024;
    const int size = N * sizeof(float);
    
    // Host vectors
    float *h_a, *h_b, *h_c;
    // Device vectors
    float *d_a, *d_b, *d_c;
    
    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);
    
    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_a[i] = i * 1.0f;
        h_b[i] = i * 2.0f;
    }
    
    // Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    
    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    // Launch kernel with 1D grid
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    printf("CUDA Setup Test - Vector Addition\n");
    printf("Vector size: %d elements\n", N);
    printf("Grid size: %d blocks\n", blocksPerGrid);
    printf("Block size: %d threads\n", threadsPerBlock);
    printf("\nLaunching kernel...\n");
    
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    
    // Check for kernel errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    // Verify results
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (abs(h_c[i] - (h_a[i] + h_b[i])) > 1e-5) {
            success = false;
            break;
        }
    }
    
    if (success) {
        printf("SUCCESS: Vector addition completed correctly!\n");
        printf("Sample results:\n");
        for (int i = 0; i < 5; i++) {
            printf("  %.1f + %.1f = %.1f\n", h_a[i], h_b[i], h_c[i]);
        }
    } else {
        printf("ERROR: Vector addition failed!\n");
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("\nGPU Information:\n");
    printf("  Device name: %s\n", prop.name);
    printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("  Total global memory: %.1f MB\n", prop.totalGlobalMem / 1024.0 / 1024.0);
    printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
    
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    
    printf("\nCUDA test completed successfully!\n");
    return 0;
}
