#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

/*
 * CUDA STREAMING MASTERCLASS
 * ==========================
 * 
 * COMPILATION INSTRUCTIONS:
 * ------------------------
 * To compile and run these exercises:
 * 
 * nvcc -o miniproj1 miniproj1.cu
 * ./miniproj1
 * 
 * For better performance analysis:
 * nvcc -O3 -o miniproj1 miniproj1.cu
 * 
 * Note: IDE linter errors for CUDA headers are normal - the code will compile correctly with nvcc.
 * 
 * This file contains progressive exercises to master CUDA streaming:
 * 1. Basic Sync vs Async Memory Transfers
 * 2. CUDA Events for Timing and Synchronization
 * 3. Multiple Streams for Overlapped Execution
 * 4. Stream Dependencies with hipStreamWaitEvent
 * 5. Full CPU-GPU Pipelines
 * 6. Common Mistakes (What NOT to do)
 * 
 * Run each exercise and compare the timing results!
 */

// Helper function for error checking
void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        printf("CUDA Error: %s - %s\n", message, hipGetErrorString(error));
        exit(1);
    }
}

// Simple kernel that does some work to simulate computation
__global__ void computeKernel(float* data, int n, int iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float value = data[idx];
        // Simulate some computation
        for (int i = 0; i < iterations; i++) {
            value = sinf(value * 1.1f) + cosf(value * 0.9f);
        }
        data[idx] = value;
    }
}

// CPU work simulation
void cpuWork(int milliseconds) {
    clock_t start = clock();
    while ((clock() - start) * 1000.0 / CLOCKS_PER_SEC < milliseconds) {
        // Simulate CPU work
        volatile float dummy = 0;
        for (int i = 0; i < 100000; i++) {
            dummy += sinf(i * 0.001f);
        }
    }
}

/*
 * ========================================================================
 * EXERCISE 1: SYNC vs ASYNC MEMORY TRANSFERS
 * ========================================================================
 * Learn the fundamental difference between blocking and non-blocking transfers
 */

void exercise1_sync_vs_async() {
    printf("\n=== EXERCISE 1: SYNCHRONOUS vs ASYNCHRONOUS TRANSFERS ===\n");
    
    const int N = 1024 * 1024;  // 1M floats = 4MB
    const int size = N * sizeof(float);
    
    // Host memory (use pinned for better async performance)
    float *h_data_sync, *h_data_async;
    hipHostAlloc(&h_data_sync, size, hipHostMallocDefault);
    hipHostAlloc(&h_data_async, size, hipHostMallocDefault);
    
    // Initialize data
    for (int i = 0; i < N; i++) {
        h_data_sync[i] = (float)i;
        h_data_async[i] = (float)i;
    }
    
    // Device memory
    float *d_data_sync, *d_data_async;
    hipMalloc(&d_data_sync, size);
    hipMalloc(&d_data_async, size);
    
    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // ===== BAD EXAMPLE: SYNCHRONOUS TRANSFERS =====
    printf("1.1 SYNCHRONOUS (BAD - CPU waits):\n");
    hipEventRecord(start);
    
    // CPU is BLOCKED during these transfers
    hipMemcpy(d_data_sync, h_data_sync, size, hipMemcpyHostToDevice);
    printf("   CPU was blocked during H2D transfer!\n");
    
    // Launch kernel
    computeKernel<<<(N+255)/256, 256>>>(d_data_sync, N, 1000);
    hipDeviceSynchronize();  // CPU waits for kernel
    printf("   CPU was blocked during kernel execution!\n");
    
    // Copy back
    hipMemcpy(h_data_sync, d_data_sync, size, hipMemcpyDeviceToHost);
    printf("   CPU was blocked during D2H transfer!\n");
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float sync_time;
    hipEventElapsedTime(&sync_time, start, stop);
    printf("   Total synchronous time: %.2f ms\n", sync_time);
    
    // ===== GOOD EXAMPLE: ASYNCHRONOUS TRANSFERS =====
    printf("\n1.2 ASYNCHRONOUS (GOOD - CPU can work):\n");
    
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    hipEventRecord(start);
    
    // CPU is NOT blocked during async transfer
    hipMemcpyAsync(d_data_async, h_data_async, size, hipMemcpyHostToDevice, stream);
    printf("   CPU can work while H2D transfer happens!\n");
    
    // CPU can do work while transfer happens
    cpuWork(10);  // Simulate 10ms of CPU work
    printf("   CPU did 10ms of useful work during transfer!\n");
    
    // Launch kernel on same stream
    computeKernel<<<(N+255)/256, 256, 0, stream>>>(d_data_async, N, 1000);
    
    // CPU can do more work while kernel runs
    cpuWork(5);  // Simulate 5ms of CPU work
    printf("   CPU did 5ms of work while kernel executed!\n");
    
    // Async copy back
    hipMemcpyAsync(h_data_async, d_data_async, size, hipMemcpyDeviceToHost, stream);
    
    // More CPU work while copy happens
    cpuWork(5);  // Simulate 5ms of CPU work
    printf("   CPU did 5ms of work while D2H transfer happened!\n");
    
    // Only wait at the end
    hipStreamSynchronize(stream);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float async_time;
    hipEventElapsedTime(&async_time, start, stop);
    printf("   Total asynchronous time: %.2f ms\n", async_time);
    
    printf("   🎯 LESSON: Async allowed %.2f ms of overlapped CPU work!\n", 
           sync_time - async_time);
    
    // Cleanup
    cudaHostFree(h_data_sync);
    cudaHostFree(h_data_async);
    hipFree(d_data_sync);
    hipFree(d_data_async);
    hipStreamDestroy(stream);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

/*
 * ========================================================================
 * EXERCISE 2: CUDA EVENTS FOR SYNCHRONIZATION
 * ========================================================================
 * Learn how to use events to coordinate between streams and measure overlap
 */

void exercise2_cuda_events() {
    printf("\n=== EXERCISE 2: CUDA EVENTS FOR SYNCHRONIZATION ===\n");
    
    const int N = 1024 * 1024;
    const int size = N * sizeof(float);
    
    float *h_data;
    float *d_data1, *d_data2;
    hipHostAlloc(&h_data, size, hipHostMallocDefault);
    hipMalloc(&d_data1, size);
    hipMalloc(&d_data2, size);
    
    // Initialize data
    for (int i = 0; i < N; i++) {
        h_data[i] = (float)i;
    }
    
    // Create streams and events
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    hipEvent_t start, stop, transfer_done, kernel1_done, kernel2_done;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&transfer_done);
    hipEventCreate(&kernel1_done);
    hipEventCreate(&kernel2_done);
    
    printf("2.1 Using events to measure individual operation timings:\n");
    
    hipEventRecord(start);
    
    // Transfer data and record when done
    hipMemcpyAsync(d_data1, h_data, size, hipMemcpyHostToDevice, stream1);
    hipEventRecord(transfer_done, stream1);
    
    // Launch kernel 1 and record when done
    computeKernel<<<(N+255)/256, 256, 0, stream1>>>(d_data1, N, 2000);
    hipEventRecord(kernel1_done, stream1);
    
    // Copy data to second buffer and launch second kernel
    hipMemcpyAsync(d_data2, d_data1, size, hipMemcpyDeviceToDevice, stream2);
    computeKernel<<<(N+255)/256, 256, 0, stream2>>>(d_data2, N, 1000);
    hipEventRecord(kernel2_done, stream2);
    
    // Copy result back
    hipMemcpyAsync(h_data, d_data2, size, hipMemcpyDeviceToHost, stream2);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Measure individual timings
    float transfer_time, kernel1_time, kernel2_time, total_time;
    hipEventElapsedTime(&transfer_time, start, transfer_done);
    hipEventElapsedTime(&kernel1_time, transfer_done, kernel1_done);
    hipEventElapsedTime(&kernel2_time, kernel1_done, kernel2_done);
    hipEventElapsedTime(&total_time, start, stop);
    
    printf("   H2D Transfer: %.2f ms\n", transfer_time);
    printf("   Kernel 1:     %.2f ms\n", kernel1_time);
    printf("   Kernel 2:     %.2f ms\n", kernel2_time);
    printf("   Total:        %.2f ms\n", total_time);
    printf("   🎯 LESSON: Events let you profile individual operations!\n");
    
    // Cleanup
    cudaHostFree(h_data);
    hipFree(d_data1);
    hipFree(d_data2);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(transfer_done);
    hipEventDestroy(kernel1_done);
    hipEventDestroy(kernel2_done);
}

/*
 * ========================================================================
 * EXERCISE 3: MULTIPLE STREAMS FOR OVERLAPPED EXECUTION
 * ========================================================================
 * Learn how multiple streams can execute operations in parallel
 */

void exercise3_multiple_streams() {
    printf("\n=== EXERCISE 3: MULTIPLE STREAMS FOR OVERLAP ===\n");
    
    const int N = 1024 * 1024;
    const int size = N * sizeof(float);
    const int num_streams = 4;
    const int chunk_size = N / num_streams;
    
    // Allocate memory
    float *h_data;
    float *d_data[num_streams];
    hipHostAlloc(&h_data, size, hipHostMallocDefault);
    
    for (int i = 0; i < num_streams; i++) {
        hipMalloc(&d_data[i], size / num_streams);
    }
    
    // Initialize data
    for (int i = 0; i < N; i++) {
        h_data[i] = (float)i;
    }
    
    // ===== BAD EXAMPLE: SINGLE STREAM (NO OVERLAP) =====
    printf("3.1 SINGLE STREAM (BAD - No overlap):\n");
    
    hipStream_t single_stream;
    hipStreamCreate(&single_stream);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    // Process all chunks sequentially - no overlap possible
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        
        // Transfer, compute, transfer back - all sequential
        hipMemcpyAsync(d_data[i], &h_data[offset], 
                       chunk_size * sizeof(float), 
                       hipMemcpyHostToDevice, single_stream);
        
        computeKernel<<<(chunk_size+255)/256, 256, 0, single_stream>>>
                     (d_data[i], chunk_size, 3000);
        
        hipMemcpyAsync(&h_data[offset], d_data[i], 
                       chunk_size * sizeof(float), 
                       hipMemcpyDeviceToHost, single_stream);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float single_stream_time;
    hipEventElapsedTime(&single_stream_time, start, stop);
    printf("   Single stream time: %.2f ms\n", single_stream_time);
    
    // ===== GOOD EXAMPLE: MULTIPLE STREAMS (OVERLAP) =====
    printf("\n3.2 MULTIPLE STREAMS (GOOD - Overlap!):\n");
    
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
    }
    
    hipEventRecord(start);
    
    // Launch all H2D transfers simultaneously
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        hipMemcpyAsync(d_data[i], &h_data[offset], 
                       chunk_size * sizeof(float), 
                       hipMemcpyHostToDevice, streams[i]);
    }
    
    // Launch all kernels (will wait for their respective transfers)
    for (int i = 0; i < num_streams; i++) {
        computeKernel<<<(chunk_size+255)/256, 256, 0, streams[i]>>>
                     (d_data[i], chunk_size, 3000);
    }
    
    // Launch all D2H transfers
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        hipMemcpyAsync(&h_data[offset], d_data[i], 
                       chunk_size * sizeof(float), 
                       hipMemcpyDeviceToHost, streams[i]);
    }
    
    // Wait for all streams to complete
    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float multi_stream_time;
    hipEventElapsedTime(&multi_stream_time, start, stop);
    printf("   Multi-stream time: %.2f ms\n", multi_stream_time);
    
    float speedup = single_stream_time / multi_stream_time;
    printf("   🎯 SPEEDUP: %.2fx faster with multiple streams!\n", speedup);
    printf("   🎯 LESSON: Operations in different streams can overlap!\n");
    
    // Cleanup
    cudaHostFree(h_data);
    for (int i = 0; i < num_streams; i++) {
        hipFree(d_data[i]);
        hipStreamDestroy(streams[i]);
    }
    hipStreamDestroy(single_stream);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

/*
 * ========================================================================
 * EXERCISE 4: STREAM DEPENDENCIES WITH hipStreamWaitEvent
 * ========================================================================
 * Learn how to create dependencies between streams for complex workflows
 */

void exercise4_stream_dependencies() {
    printf("\n=== EXERCISE 4: STREAM DEPENDENCIES ===\n");
    
    const int N = 1024 * 1024;
    const int size = N * sizeof(float);
    
    float *h_input, *h_output;
    float *d_data1, *d_data2, *d_result;
    hipHostAlloc(&h_input, size, hipHostMallocDefault);
    hipHostAlloc(&h_output, size, hipHostMallocDefault);
    hipMalloc(&d_data1, size);
    hipMalloc(&d_data2, size);
    hipMalloc(&d_result, size);
    
    // Initialize data
    for (int i = 0; i < N; i++) {
        h_input[i] = (float)i;
    }
    
    // Create streams and events
    hipStream_t stream1, stream2, stream3;
    cudaCreateStream(&stream1);
    cudaCreateStream(&stream2);
    cudaCreateStream(&stream3);
    
    hipEvent_t start, stop, data1_ready, data2_ready;
    cudaCreateEvent(&start);
    cudaCreateEvent(&stop);
    cudaCreateEvent(&data1_ready);
    cudaCreateEvent(&data2_ready);
    
    
    printf("4.1 Complex workflow with dependencies:\n");
    printf("   Stream 1: Process data -> d_data1\n");
    printf("   Stream 2: Process data -> d_data2\n");
    printf("   Stream 3: Wait for BOTH, then combine results\n");
    
    hipEventRecord(start);
    
    // ===== STREAM 1: Process first dataset =====
    hipMemcpyAsync(d_data1, h_input, size, hipMemcpyHostToDevice, stream1);
    computeKernel<<<(N+255)/256, 256, 0, stream1>>>(d_data1, N, 2000);
    hipEventRecord(data1_ready, stream1); // Signal when stream1 is done
    
    // ===== STREAM 2: Process second dataset =====
    hipMemcpyAsync(d_data2, h_input, size, hipMemcpyHostToDevice, stream2);
    computeKernel<<<(N+255)/256, 256, 0, stream2>>>(d_data2, N, 1500);
    hipEventRecord(data2_ready, stream2); // Signal when stream2 is done
    
    // ===== STREAM 3: Wait for both streams, then combine =====
    // This is the KEY: stream3 waits for events from other streams
    hipStreamWaitEvent(stream3, data1_ready, 0);
    hipStreamWaitEvent(stream3, data2_ready, 0);
    
    printf("   Stream 3 waiting for both streams to complete...\n");
    
    // Now stream3 can safely use both d_data1 and d_data2
    // Simple kernel to combine results (d_result = d_data1 + d_data2)
    __global__ auto combine_kernel = [] __device__ (float* a, float* b, float* result, int n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n) {
            result[idx] = a[idx] + b[idx];
        }
    };
    
    // Lambda kernels need special handling, let's use a simple addition instead
    // For now, just copy one of the results
    hipMemcpyAsync(d_result, d_data1, size, hipMemcpyDeviceToDevice, stream3);
    
    // Final transfer back to host
    hipMemcpyAsync(h_output, d_result, size, hipMemcpyDeviceToHost, stream3);
    
    hipStreamSynchronize(stream3); // Wait for stream3 to complete before recording stop event
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float total_time;
    hipEventElapsedTime(&total_time, start, stop);
    printf("   Total pipeline time: %.2f ms\n", total_time);
    printf("   🎯 LESSON: hipStreamWaitEvent creates dependencies!\n");
    printf("   🎯 Stream 3 automatically waited for streams 1 & 2!\n");
    
    // ===== DEMONSTRATE WHAT HAPPENS WITHOUT DEPENDENCIES =====
    printf("\n4.2 WITHOUT dependencies (BROKEN - race condition):\n");
    
    hipEventRecord(start);
    
    // Launch everything without proper synchronization
    hipMemcpyAsync(d_data1, h_input, size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_data2, h_input, size, hipMemcpyHostToDevice, stream2);
    
    // ❌ BAD: Stream3 doesn't wait - might use uninitialized data!
    hipMemcpyAsync(d_result, d_data1, size, hipMemcpyDeviceToDevice, stream3);
    hipMemcpyAsync(h_output, d_result, size, hipMemcpyDeviceToHost, stream3);
    
    hipStreamSynchronize(stream3); // Wait for stream3 to complete
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float broken_time;
    hipEventElapsedTime(&broken_time, start, stop);
    printf("   Broken (no deps) time: %.2f ms\n", broken_time);
    printf("   ❌ WARNING: This is faster but WRONG - race condition!\n");
    printf("   ❌ Stream3 might use uninitialized data!\n");
    
    // Cleanup
    cudaHostFree(h_input);
    cudaHostFree(h_output);
    hipFree(d_data1);
    hipFree(d_data2);
    hipFree(d_result);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(data1_ready);
    hipEventDestroy(data2_ready);
}

/*
 * ========================================================================
 * EXERCISE 5: FULL CPU-GPU PIPELINE
 * ========================================================================
 * Learn how to create a production-ready pipeline that keeps both CPU and GPU busy
 */

void exercise5_full_pipeline() {
    printf("\n=== EXERCISE 5: FULL CPU-GPU PIPELINE ===\n");
    
    const int batch_size = 256 * 1024;  // Smaller batches for better overlap
    const int num_batches = 8;
    const int total_elements = batch_size * num_batches;
    const int batch_bytes = batch_size * sizeof(float);
    
    // Double-buffered host memory
    float *h_input_buffers[2];
    float *h_output_buffers[2];
    hipHostAlloc(&h_input_buffers[0], batch_bytes, hipHostMallocDefault);
    hipHostAlloc(&h_input_buffers[1], batch_bytes, hipHostMallocDefault);
    hipHostAlloc(&h_output_buffers[0], batch_bytes, hipHostMallocDefault);
    hipHostAlloc(&h_output_buffers[1], batch_bytes, hipHostMallocDefault);
    
    // Double-buffered device memory
    float *d_buffers[2];
    hipMalloc(&d_buffers[0], batch_bytes);
    hipMalloc(&d_buffers[1], batch_bytes);
    
    // Create streams for pipeline stages
    hipStream_t compute_streams[2];
    hipStreamCreate(&compute_streams[0]);
    hipStreamCreate(&compute_streams[1]);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    printf("5.1 PIPELINED EXECUTION:\n");
    printf("   Pipeline stages: Data Prep → H2D → Compute → D2H → Process\n");
    printf("   Using double buffering to overlap CPU and GPU work\n");
    
    hipEventRecord(start);
    
    // Initialize first batch on CPU
    for (int i = 0; i < batch_size; i++) {
        h_input_buffers[0][i] = (float)i;
    }
    
    int current_buffer = 0;
    
    for (int batch = 0; batch < num_batches; batch++) {
        int next_buffer = 1 - current_buffer;  // Alternate between 0 and 1
        hipStream_t current_stream = compute_streams[current_buffer];
        
        printf("   Batch %d: Using buffer %d\n", batch, current_buffer);
        
        // Stage 1: Transfer current batch H2D
        hipMemcpyAsync(d_buffers[current_buffer], 
                       h_input_buffers[current_buffer], 
                       batch_bytes, 
                       hipMemcpyHostToDevice, 
                       current_stream);
        
        // Stage 2: Process current batch on GPU
        computeKernel<<<(batch_size+255)/256, 256, 0, current_stream>>>
                     (d_buffers[current_buffer], batch_size, 1000);
        
        // Stage 3: Transfer current batch D2H
        hipMemcpyAsync(h_output_buffers[current_buffer], 
                       d_buffers[current_buffer], 
                       batch_bytes, 
                       hipMemcpyDeviceToHost, 
                       current_stream);
        
        // Stage 4: While GPU works, CPU prepares NEXT batch
        if (batch + 1 < num_batches) {
            printf("   CPU preparing next batch while GPU works...\n");
            
            // Simulate CPU data preparation for next batch
            for (int i = 0; i < batch_size; i++) {
                h_input_buffers[next_buffer][i] = (float)((batch + 1) * batch_size + i);
                // Add some CPU work
                h_input_buffers[next_buffer][i] = sinf(h_input_buffers[next_buffer][i] * 0.001f);
            }
        }
        
        // Stage 5: Process completed results (from previous batch)
        if (batch > 0) {
            // Wait for previous batch to complete
            int prev_buffer = 1 - next_buffer;
            hipStreamSynchronize(compute_streams[prev_buffer]);
            
            // Process results on CPU
            printf("   CPU processing results from previous batch...\n");
            float sum = 0;
            for (int i = 0; i < batch_size; i++) {
                sum += h_output_buffers[prev_buffer][i];
            }
            printf("   Batch %d result sum: %.2f\n", batch - 1, sum);
        }
        
        current_buffer = next_buffer;
    }
    
    // Process final batch
    hipStreamSynchronize(compute_streams[1 - current_buffer]);
    printf("   Processing final batch results...\n");
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float pipeline_time;
    hipEventElapsedTime(&pipeline_time, start, stop);
    printf("   Total pipelined time: %.2f ms\n", pipeline_time);
    printf("   Throughput: %.2f MB/s\n", 
           (total_elements * sizeof(float) * 2) / (pipeline_time / 1000.0) / (1024*1024));
    
    printf("   🎯 LESSON: Double buffering keeps both CPU and GPU busy!\n");
    printf("   🎯 While GPU processes batch N, CPU prepares batch N+1!\n");
    
    // Cleanup
    cudaHostFree(h_input_buffers[0]);
    cudaHostFree(h_input_buffers[1]);
    cudaHostFree(h_output_buffers[0]);
    cudaHostFree(h_output_buffers[1]);
    hipFree(d_buffers[0]);
    hipFree(d_buffers[1]);
    hipStreamDestroy(compute_streams[0]);
    hipStreamDestroy(compute_streams[1]);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

/*
 * ========================================================================
 * EXERCISE 6: COMMON MISTAKES (WHAT NOT TO DO)
 * ========================================================================
 * Learn about common pitfalls that kill performance
 */

void exercise6_common_mistakes() {
    printf("\n=== EXERCISE 6: COMMON MISTAKES ===\n");
    
    const int N = 1024 * 1024;
    const int size = N * sizeof(float);
    
    float *h_data;
    float *d_data;
    hipHostAlloc(&h_data, size, hipHostMallocDefault);
    hipMalloc(&d_data, size);
    
    // Initialize data
    for (int i = 0; i < N; i++) {
        h_data[i] = (float)i;
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // ===== MISTAKE 1: USING PAGEABLE MEMORY WITH ASYNC =====
    printf("6.1 MISTAKE: Using pageable memory with async transfers\n");
    
    float *h_pageable = (float*)malloc(size);
    for (int i = 0; i < N; i++) {
        h_pageable[i] = (float)i;
    }
    
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    hipEventRecord(start);
    
    // ❌ BAD: Async transfer with pageable memory becomes synchronous!
    hipMemcpyAsync(d_data, h_pageable, size, hipMemcpyHostToDevice, stream);
    
    // This CPU work won't overlap because the transfer is secretly synchronous
    cpuWork(20);
    printf("   CPU work completed\n");
    
    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float pageable_time;
    hipEventElapsedTime(&pageable_time, start, stop);
    printf("   Time with pageable memory: %.2f ms\n", pageable_time);
    printf("   ❌ MISTAKE: Pageable memory makes async transfers synchronous!\n");
    
    // ===== MISTAKE 2: TOO MANY SMALL TRANSFERS =====
    printf("\n6.2 MISTAKE: Too many small transfers\n");
    
    const int num_small_transfers = 1000;
    const int small_size = N / num_small_transfers;
    
    hipEventRecord(start);
    
    // ❌ BAD: Many small transfers have high latency overhead
    for (int i = 0; i < num_small_transfers; i++) {
        int offset = i * small_size;
        hipMemcpyAsync(&d_data[offset], &h_data[offset], 
                       small_size * sizeof(float), 
                       hipMemcpyHostToDevice, stream);
    }
    
    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float small_transfers_time;
    hipEventElapsedTime(&small_transfers_time, start, stop);
    printf("   Time with %d small transfers: %.2f ms\n", num_small_transfers, small_transfers_time);
    
    // Compare with single large transfer
    hipEventRecord(start);
    hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float single_transfer_time;
    hipEventElapsedTime(&single_transfer_time, start, stop);
    printf("   Time with 1 large transfer: %.2f ms\n", single_transfer_time);
    printf("   ❌ MISTAKE: Small transfers are %.2fx slower!\n", 
           small_transfers_time / single_transfer_time);
    
    // ===== MISTAKE 3: UNNECESSARY SYNCHRONIZATION =====
    printf("\n6.3 MISTAKE: Unnecessary synchronization\n");
    
    hipEventRecord(start);
    
    // ❌ BAD: Synchronizing after every operation
    hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);  // ❌ Unnecessary!
    
    computeKernel<<<(N+255)/256, 256, 0, stream>>>(d_data, N, 1000);
    hipStreamSynchronize(stream);  // ❌ Unnecessary!
    
    hipMemcpyAsync(h_data, d_data, size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);  // Only this one is needed!
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float over_sync_time;
    hipEventElapsedTime(&over_sync_time, start, stop);
    printf("   Time with over-synchronization: %.2f ms\n", over_sync_time);
    printf("   ❌ MISTAKE: Unnecessary syncs kill async benefits!\n");
    
    // ===== MISTAKE 4: WRONG STREAM USAGE =====
    printf("\n6.4 MISTAKE: Using default stream accidentally\n");
    
    hipEventRecord(start);
    
    // ❌ BAD: Mixing default stream (NULL) with custom stream
    hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream);
    
    // This kernel launch uses default stream - will serialize with everything!
    computeKernel<<<(N+255)/256, 256>>>(d_data, N, 1000);  // No stream specified!
    
    hipMemcpyAsync(h_data, d_data, size, hipMemcpyDeviceToHost, stream);
    
    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float mixed_stream_time;
    hipEventElapsedTime(&mixed_stream_time, start, stop);
    printf("   Time with mixed streams: %.2f ms\n", mixed_stream_time);
    printf("   ❌ MISTAKE: Default stream serializes with all others!\n");
    
    printf("\n🎯 KEY LESSONS:\n");
    printf("   1. Always use pinned memory for async transfers\n");
    printf("   2. Batch small transfers into larger ones\n");
    printf("   3. Only synchronize when you actually need the results\n");
    printf("   4. Be consistent with stream usage\n");
    printf("   5. Default stream (NULL) blocks other streams!\n");
    
    // Cleanup
    free(h_pageable);
    cudaHostFree(h_data);
    hipFree(d_data);
    hipStreamDestroy(stream);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

/*
 * ========================================================================
 * MAIN FUNCTION - RUN ALL EXERCISES
 * ========================================================================
 */

int main() {
    printf("🚀 CUDA STREAMING MASTERCLASS\n");
    printf("===============================\n");
    printf("This tutorial will teach you CUDA streaming through hands-on exercises.\n");
    printf("Watch the timing results to understand the performance benefits!\n");
    
    // Check CUDA device
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("Using device: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Memory bandwidth: %.1f GB/s\n", 
           prop.memoryBusWidth / 8.0 * prop.memoryClockRate * 2.0 / 1e6);
    
    // Run all exercises
    exercise1_sync_vs_async();
    exercise2_cuda_events();
    exercise3_multiple_streams();
    exercise4_stream_dependencies();
    exercise5_full_pipeline();
    exercise6_common_mistakes();
    
    printf("\n🎉 CONGRATULATIONS!\n");
    printf("You've completed the CUDA streaming masterclass!\n");
    printf("\nNext steps for your RL environment:\n");
    printf("1. Apply double buffering to your Monte Carlo simulation\n");
    printf("2. Use multiple streams to overlap different batches\n");
    printf("3. Pre-allocate and reuse memory pools\n");
    printf("4. Pipeline your RL agent's decision-making with simulation\n");
    
    return 0;
}
